#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

__global__ void add(int n, float* x, float* y, float* z){
    int index = threadIdx.x;
    int stride = blockDim.x;

    for(int i=index; i<n; i+= stride){
        z[i] = x[i] + y[i];
    }
}

int main(){
int N = 1 <<20;
float *x, *y, *z;


hipMallocManaged(&x, N*sizeof(float));
hipMallocManaged(&y, N*sizeof(float));
hipMallocManaged(&z, N*sizeof(float));

for(int i=0; i<N;i++){
    x[i]=float(3.0);
    y[i]=float(2.0);
}

  char *prefetch = getenv("__PREFETCH");
  if (prefetch == NULL || strcmp(prefetch, "off") != 0) {
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(x, N*sizeof(float), device, NULL);
    hipMemPrefetchAsync(y, N*sizeof(float), device, NULL);
    hipMemPrefetchAsync(z, N*sizeof(float), device, NULL);
  }
 


add<<<1, 256>>>(N, x,y,z);
hipDeviceSynchronize();

std::cout<<z[3]<<std::endl;
hipFree(x);
hipFree(y);
hipFree(z);




}