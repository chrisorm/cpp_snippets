
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void cuda_hello_world(){
    printf("Hello from my GPU\n");
}

int main(){
    cuda_hello_world<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}